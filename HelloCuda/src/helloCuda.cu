#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello(void){
    printf("hello CUDA %d!\n", threadIdx.x);
}

int main(void){
    hello<<<1,8>>>();
	hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    return 0;
}