#include "hip/hip_runtime.h"
#include "trapezoidal/trapezoidal_gpu.h"
#include "trapezoidal/trapezoidal_cpu.h"

#include "hip/hip_runtime.h"
#include ""

__host__ __device__  double func(double x)
{
    return (x * x);
}

__host__ __device__ double Trapezoidal(long i, double h)
{
    double a = START + i * h;
    double b = a + h;
    return (func(a) + func(b)) * h / 2;
}

void serialTrapezoidal(double &serial_res, double h, DS_timer&timer, int mode)
{
    timer.onTimer(mode);

    for (long i = 0; i < SECTION; ++i)
        serial_res += Trapezoidal(i, h);

    timer.offTimer(mode);
}

void ompTrapezoidal(double &omp_res, double h, DS_timer&timer, int mode)
{
    timer.onTimer(mode);

    #pragma omp parallel num_threads(THREAD_NUM) reduction(+ : omp_res)
    {
        #pragma omp for
        for (long i = 0; i < SECTION; ++i)
            omp_res += Trapezoidal(i, h);
    }

    timer.offTimer(mode);
}

__global__ void trapezoidal(double h, double*d_res)
{
    long idx = long(blockIdx.x) * long(blockDim.x) + threadIdx.x;

	if(idx < SECTION){
        double a = START + idx * h;
        double b = a + h;
        double sum = (a*a + b*b) * h / 2;
	    atomicAdd(d_res, sum);
    }
}

__global__ void trapezoidalOptimizing(double h, double*d_res)
{
    long tid = long(blockIdx.x) * long(blockDim.x) + threadIdx.x;

	__shared__ double localVal[BLOCK_SIZE];
	localVal[threadIdx.x] = 0;

	if (tid >= SECTION)
		return;

	localVal[threadIdx.x] = Trapezoidal(tid, h);

	__syncthreads();

	// reduction here
	long offset = BLOCK_SIZE / 2;

	while (offset > 0) {
		if (threadIdx.x < offset) {
			localVal[threadIdx.x] += localVal[threadIdx.x + offset];
		}
		offset /= 2;

		__syncthreads();
	}

	if (threadIdx.x == 0) {
		atomicAdd(d_res, localVal[0]);
	}
}

void kernelCall(double&cuda_res, double h, DS_timer&timer, int mode){
    dim3 dimGrid(ceil(SECTION / (float)BLOCK_SIZE));
    double*d_res;

    timer.onTimer(mode+2);
    hipMalloc(&d_res, sizeof(double));
    hipMemset(d_res, 0, sizeof(double));
    timer.offTimer(mode+2);

    timer.onTimer(mode);

    switch(mode)
    {
    case CUDA_BASIC:
	    trapezoidal<<<dimGrid, BLOCK_SIZE>>> (h, d_res);
	    break;
    case CUDA_OPTIMIZING:
        trapezoidalOptimizing<<<dimGrid, BLOCK_SIZE>>> (h, d_res);
        break;
    default:
        break;
    }

    hipDeviceSynchronize();
    
    timer.offTimer(mode);

    timer.onTimer(mode+2);
    hipMemcpy(&cuda_res, d_res, sizeof(double), hipMemcpyDeviceToHost);
    timer.offTimer(mode+2);
}