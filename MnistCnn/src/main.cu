#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#include "mnist.h"
#include "layer.h"

#include "hip/hip_runtime.h"
#include ""

mnist_data *train_set, *test_set;
unsigned int train_cnt, test_cnt;

Layer l_input = Layer(0, 0, 28 * 28);
Layer l_c = Layer(5 * 5, 6, 24 * 24 * 6);
Layer l_s = Layer(4 * 4, 1, 6 * 6 * 6);
Layer l_f = Layer(6 * 6 * 6, 10, 10);

void loadData();
void learn();
unsigned int classify(double data[28][28]);
void test();
double forward_propagation(double data[28][28]);
double back_propagation();

int main(int argc, char const *argv[])
{
	srand(time(NULL));
	loadData();
	learn();
	test();
	return 0;
}

void loadData()
{
	mnist_load("data/train-images.idx3-ubyte", "data/train-labels.idx1-ubyte", &train_set, &train_cnt);
	mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte", &test_set, &test_cnt);
}

double forward_propagation(double data[28][28])
{
	float input[28][28];

	for (int i = 0; i < 28; ++i) {
		for (int j = 0; j < 28; ++j) {
			input[i][j] = data[i][j];
		}
	}

	l_input.clear();
	l_c.clear();
	l_s.clear();
	l_f.clear();

	clock_t start, end;
	start = clock();

	l_input.setOutput((float *)input);
	
	fp_preact_c<<<64, 64>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c.preact, (float (*)[5][5])l_c.weight);
	fp_bias_c<<<64, 64>>>((float (*)[24][24])l_c.preact, l_c.bias);
	apply_activ_func<<<64, 64>>>(l_c.preact, l_c.output, l_c.O);

	fp_preact_s<<<64, 64>>>((float (*)[24][24])l_c.output, (float (*)[6][6])l_s.preact, (float (*)[4][4])l_s.weight);
	fp_bias_s<<<64, 64>>>((float (*)[6][6])l_s.preact, l_s.bias);
	apply_activ_func<<<64, 64>>>(l_s.preact, l_s.output, l_s.O);

	fp_preact_f<<<64, 64>>>((float (*)[6][6])l_s.output, l_f.preact, (float (*)[6][6][6])l_f.weight);
	fp_bias_f<<<64, 64>>>(l_f.preact, l_f.bias);
	apply_activ_func<<<64, 64>>>(l_f.preact, l_f.output, l_f.O);
	
	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

double back_propagation()
{
	clock_t start, end;

	start = clock();

	bp_weight_f<<<64, 64>>>((float(*)[6][6][6])l_f.d_weight, l_f.d_preact, (float(*)[6][6])l_s.output);
	bp_bias_f<<<64, 64>>>(l_f.bias, l_f.d_preact);

	bp_output_s<<<64, 64>>>((float(*)[6][6])l_s.d_output, (float(*)[6][6][6])l_f.weight, l_f.d_preact);
	bp_preact_s<<<64, 64>>>((float(*)[6][6])l_s.d_preact, (float(*)[6][6])l_s.d_output, (float(*)[6][6])l_s.preact);
	bp_weight_s<<<64, 64>>>((float(*)[4][4])l_s.d_weight, (float(*)[6][6])l_s.d_preact, (float(*)[24][24])l_c.output);
	bp_bias_s<<<64, 64>>>(l_s.bias, (float(*)[6][6])l_s.d_preact);

	bp_output_c<<<64, 64>>>((float(*)[24][24])l_c.d_output, (float(*)[4][4])l_s.weight, (float(*)[6][6])l_s.d_preact);
	bp_preact_c<<<64, 64>>>((float(*)[24][24])l_c.d_preact, (float(*)[24][24])l_c.d_output, (float(*)[24][24])l_c.preact);
	bp_weight_c<<<64, 64>>>((float(*)[5][5])l_c.d_weight, (float(*)[24][24])l_c.d_preact, (float(*)[28])l_input.output);
	bp_bias_c<<<64, 64>>>(l_c.bias, (float(*)[24][24])l_c.d_preact);

	update_grad<<<64, 64>>>(l_f.weight, l_f.d_weight, l_f.M * l_f.N);
	update_grad<<<64, 64>>>(l_s.weight, l_s.d_weight, l_s.M * l_s.N);
	update_grad<<<64, 64>>>(l_c.weight, l_c.d_weight, l_c.M * l_c.N);

	end = clock();
	return ((double)(end - start)) / CLOCKS_PER_SEC;
}

void learn()
{
	hipblasHandle_t blas;
	hipblasCreate(&blas);

	float err;
	int iter = 50;

	double time_taken = 0.0;

	printf("Learning\n");

	while (iter < 0 || iter-- > 0)
	{
		err = 0.0f;

		for (int i = 0; i < train_cnt; ++i)
		{
			float tmp_err;

			time_taken += forward_propagation(train_set[i].data);

			l_f.bp_clear();
			l_s.bp_clear();
			l_c.bp_clear();

			update_error<<<10, 1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);
			hipblasSnrm2(blas, 10, l_f.d_preact, 1, &tmp_err);
			err += tmp_err;

			time_taken += back_propagation();
		}

		err /= train_cnt;
		printf("error: %e, time_on_gpu: %lf\n", err, time_taken);

		if (err < threshold)
		{
			printf("Training complete, error less than threshold\n\n");
			break;
		}
	}

	printf("\n Time - %lf\n", time_taken);
}

unsigned int classify(double data[28][28])
{
	float res[10];
	unsigned int max = 0;

	forward_propagation(data);

	hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);

	for (int i = 1; i < 10; ++i)
		if (res[max] < res[i])
			max = i;

	return max;
}

void test()
{
	int error = 0;

	for (int i = 0; i < test_cnt; ++i)
		if (classify(test_set[i].data) != test_set[i].label)
			++error;

	printf("Error Rate: %.2lf%%\n", double(error) / double(test_cnt) * 100.0);
}