#include "cnn/layer.h"

// layer 생성자
Layer::Layer(int M, int N, int O)
{
    this->M = M;
    this->N = N;
    this->O = O;

    float h_bias[N];
    float h_weight[N][M];

    output = NULL;
    preact = NULL;
    bias = NULL;
    weight = NULL;

    for (int i = 0; i < N; ++i)
    {
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);

        for (int j = 0; j < M; ++j)
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
    }

    hipMalloc(&output, sizeof(float) * O);
    hipMalloc(&preact, sizeof(float) * O);
    hipMalloc(&bias, sizeof(float) * N);
    hipMalloc(&weight, sizeof(float) * M * N);

    hipMalloc(&d_output, sizeof(float) * O);
    hipMalloc(&d_preact, sizeof(float) * O);
    hipMalloc(&d_weight, sizeof(float) * M * N);

    hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// layer 소멸자
Layer::~Layer()
{
    hipFree(output);
    hipFree(preact);
    hipFree(bias);
    hipFree(weight);

    hipFree(d_output);
    hipFree(d_preact);
    hipFree(d_weight);
}

// host -> device 메모리 복사
void Layer::setOutput(float *data)
{
    hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory
void Layer::clear()
{
    hipMemset(output, 0, sizeof(float) * O);
    hipMemset(preact, 0, sizeof(float) * O);
}

void Layer::bp_clear()
{
    hipMemset(d_output, 0, sizeof(float) * O);
    hipMemset(d_preact, 0, sizeof(float) * O);
    hipMemset(d_weight, 0, sizeof(float) * M * N);
}

