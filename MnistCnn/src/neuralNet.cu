#include "hip/hip_runtime.h"
#include "cnn/neuralNet.h"

// 시그모이드 함수
__device__ float activ_func(float v)
{
    return 1 / (1 + exp(-v));
}

// 입력 값에 대하여 활성화 함수 적용
__global__ void apply_activ_func(float *input, float *output, int N)
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    for (int i = N * t_id / size; i < N * (t_id + 1) / size; ++i)
        output[i] = activ_func(input[i]);
}

// 오차 계산
__global__ void update_error(float *err, float *output, unsigned int Y, int N)
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    for (int i = N * t_id / size; i < N * (t_id + 1) / size; ++i)
        err[i] = ((Y == i ? 1.0f : 0.0f) - output[i]);
}

// 기울기 계산
__global__ void update_grad(float *output, float *grad, int N)
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    for (int i = N * t_id / size; i < N * (t_id + 1) / size; ++i)
        output[i] += dt * grad[i];
}

// 
__global__ void fp_preact_c(float input[28][28], float preact[6][24][24], float weight[6][5][5])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 5 * 5 * 6 * 24 * 24;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 5);
        int i2 = ((idx /= 5) % 5);
        int i3 = ((idx /= 5) % 6);
        int i4 = ((idx /= 6) % 24);
        int i5 = ((idx /= 24) % 24);

        atomicAdd(&preact[i3][i4][i5], weight[i3][i1][i2] * input[i4 + i1][i5 + i2]);
    }
}

// 
__global__ void fp_bias_c(float preact[6][24][24], float bias[6])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 24 * 24;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 24);
        int i3 = ((idx /= 24) % 24);

        preact[i1][i2][i3] += bias[i1];
    }
}

//
__global__ void fp_preact_s(float input[6][24][24], float preact[6][6][6], float weight[1][4][4])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 4 * 4 * 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 4);
        int i2 = ((idx /= 4) % 4);
        int i3 = ((idx /= 4) % 6);
        int i4 = ((idx /= 6) % 6);
        int i5 = ((idx /= 6) % 6);

        atomicAdd(&preact[i3][i4][i5], weight[0][i1][i2] * input[i3][i4 * 4 + i1][i5 * 4 + i2]);
    }
}

// 
__global__ void fp_bias_s(float preact[6][6][6], float bias[1])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 6);
        int i3 = ((idx /= 6) % 6);

        preact[i1][i2][i3] += bias[0];
    }
}

// 
__global__ void fp_preact_f(float input[6][6][6], float preact[10], float weight[10][6][6][6])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 10 * 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 10);
        int i2 = ((idx /= 10) % 6);
        int i3 = ((idx /= 6) % 6);
        int i4 = ((idx /= 6) % 6);

        atomicAdd(&preact[i1], weight[i1][i2][i3][i4] * input[i2][i3][i4]);
    }
}

// 
__global__ void fp_bias_f(float preact[10], float bias[10])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 10;

    for (int i = N * t_id / size; i < N * (t_id + 1) / size; ++i)
        preact[i] += bias[i];
}

//
__global__ void bp_weight_f(float d_weight[10][6][6][6], float d_preact[10], float p_output[6][6][6])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 10 * 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 10);
        int i2 = ((idx /= 10) % 6);
        int i3 = ((idx /= 6) % 6);
        int i4 = ((idx /= 6) % 6);

        d_weight[i1][i2][i3][i4] = d_preact[i1] * p_output[i2][i3][i4];
    }
}

//
__global__ void bp_bias_f(float bias[10], float d_preact[10])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 10;

    for (int idx = N * t_id / size; idx < N * (t_id + 1) / size; ++idx)
        bias[idx] += dt * d_preact[idx];
}

//
__global__ void bp_output_s(float d_output[6][6][6], float n_weight[10][6][6][6], float nd_preact[10])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 10 * 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 10);
        int i2 = ((idx /= 10) % 6);
        int i3 = ((idx /= 6) % 6);
        int i4 = ((idx /= 6) % 6);

        atomicAdd(&d_output[i2][i3][i4], n_weight[i1][i2][i3][i4] * nd_preact[i1]);
    }
}

//
__global__ void bp_preact_s(float d_preact[6][6][6], float d_output[6][6][6], float preact[6][6][6])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 6);
        int i3 = ((idx /= 6) % 6);

        float o = activ_func(preact[i1][i2][i3]);

        d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
    }
}

//
__global__ void bp_weight_s(float d_weight[1][4][4], float d_preact[6][6][6], float p_output[6][24][24])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 1 * 4 * 4 * 6 * 6 * 6;
    float d = pow(6.0f, 3.0f);

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 1);
        int i2 = ((idx /= 1) % 4);
        int i3 = ((idx /= 4) % 4);
        int i4 = ((idx /= 4) % 6);
        int i5 = ((idx /= 6) % 6);
        int i6 = ((idx /= 6) % 6);

        atomicAdd(&d_weight[i1][i2][i3], d_preact[i4][i5][i6] * p_output[i4][i5 * 4 + i2][i6 * 4 + i3]);
    }
}

//
__global__ void bp_bias_s(float bias[1], float d_preact[6][6][6])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 6 * 6;
    float d = pow(6.0f, 3.0f);

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 6);
        int i3 = ((idx /= 6) % 6);

        atomicAdd(&bias[0], dt * d_preact[i1][i2][i3] / d);
    }
}

//
__global__ void bp_output_c(float d_output[6][24][24], float n_weight[1][4][4], float nd_preact[6][6][6])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 1 * 4 * 4 * 6 * 6 * 6;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 1);
        int i2 = ((idx /= 1) % 4);
        int i3 = ((idx /= 4) % 4);
        int i4 = ((idx /= 4) % 6);
        int i5 = ((idx /= 6) % 6);
        int i6 = ((idx /= 6) % 6);

        atomicAdd(&d_output[i4][i5 * 4 + i2][i6 * 4 + i3], n_weight[i1][i2][i3] * nd_preact[i4][i5][i6]);
    }
}

//
__global__ void bp_preact_c(float d_preact[6][24][24], float d_output[6][24][24], float preact[6][24][24])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 24 * 24;

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 24);
        int i3 = ((idx /= 24) % 24);

        float o = activ_func(preact[i1][i2][i3]);

        d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
    }
}

//
__global__ void bp_weight_c(float d_weight[6][5][5], float d_preact[6][24][24], float p_output[28][28])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 5 * 5 * 24 * 24;
    float d = pow(24.0f, 2.0f);

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 5);
        int i3 = ((idx /= 5) % 5);
        int i4 = ((idx /= 5) % 24);
        int i5 = ((idx /= 24) % 24);

        atomicAdd(&d_weight[i1][i2][i3], d_preact[i1][i4][i5] * p_output[i4 + i2][i5 + i3] / d);
    }
}

//
__global__ void bp_bias_c(float bias[6], float d_preact[6][24][24])
{
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = 6 * 24 * 24;
    float d = pow(24.0f, 2.0f);

    for (int n = N * t_id / size; n < N * (t_id + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 6);
        int i2 = ((idx /= 6) % 24);
        int i3 = ((idx /= 24) % 24);

        atomicAdd(&bias[i1], dt * d_preact[i1][i2][i3] / d);
    }
}
